#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

__global__ void f(double *t, double *x, double *res) {
	int idx = threadIdx.x;
	res[idx] = sin(2 * t[idx]) + x[idx];
}

int main()
{
	int quant = 100;
	double* t0 = new double[quant];
	double* x0 = new double[quant];
	int n = 1000;
	for (int i = 0; i < quant; i++) {
		t0[i] = rand() * 100;
		x0[i] = rand() * 100;
	}
	double* x1 = x0;
	double** k = new double*[4];
	for (int i = 0; i < 4; i++)
		k[i] = new double[quant];
	for (int i = 0; i < n; i++) {
		double *k1;
		hipMalloc((void**) &k1, sizeof(int) * quant);
		f << < 1, quant >> > (t0, x0, k1);
		hipMemcpy(&k[0], k1, sizeof(int) * quant, hipMemcpyDeviceToHost);
		double *k2;
		hipMalloc((void**)&k2, sizeof(int) * quant);
		f << < 1, quant >> > (t0, x0, k2);
		hipMemcpy(&k[1], k2, sizeof(int) * quant, hipMemcpyDeviceToHost);
		double *k3;
		hipMalloc((void**) &k3, sizeof(int) * quant);
		f << <1, quant >> > (t0, x0, k3);
		hipMemcpy(&k[2], k3, sizeof(int) * quant, hipMemcpyDeviceToHost);
		double *k4;
		hipMalloc((void**) &k4, sizeof(int) * quant);
		f << <1, quant >> > (t0, x0, k4);
		hipMemcpy(&k[3], k4, sizeof(int) * quant, hipMemcpyDeviceToHost);

		for (int j = 0; j < quant; j++) {
			x0[j] = x1[j] + (k[0][j] + 2 * k[1][j] + 2 * k[2][j] + k[3][j]) / 6;
		}
	}
	for (int i = 0; i < quant; i++) {
		cout << x0[i] << " ";
	}
	return 0;
}
